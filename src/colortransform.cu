#include "hip/hip_runtime.h"
#include "cuda_methods.h"

#include <>
// CUDA runtime
#include <hip/hip_runtime.h>

// helper functions and utilities to work with CUDA
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

__global__ void rgb2yuvKernel(int *imgr,int *imgg,int *imgb,int *imgy,int *imgcb,int *imgcr, int n) {

    int r, g, b;
    int y, cb, cr;

    int index;
    index = threadIdx.x + blockIdx.x * blockDim.x;

    if (index < n){
        r = imgr[index];
        g = imgg[index];
        b = imgb[index];

        y  = (int)( 0.299*r + 0.587*g +  0.114*b);
        cb = (int)(-0.169*r - 0.331*g +  0.499*b + 128);
        cr = (int)( 0.499*r - 0.418*g - 0.0813*b + 128);

        imgy[index] = y;
        imgcb[index] = cb;
        imgcr[index] = cr;
    }
}

void rgb2yuv(int *imgr,int *imgg,int *imgb,int *imgy,int *imgcb,int *imgcr, int n){
    int *imgr_cuda, *imgg_cuda, *imgb_cuda;
    int *imgy_cuda, *imgcb_cuda, *imgcr_cuda;

    unsigned int nBytes = sizeof(int) * n;
    int threadsPerBlock = 256;
    int blocksPerGrid   = (n + threadsPerBlock - 1) / threadsPerBlock;

    hipMalloc((void **)&imgr_cuda, nBytes);
    hipMalloc((void **)&imgg_cuda, sizeof(int)*n);
    hipMalloc((void **)&imgb_cuda, sizeof(int)*n);

    hipMalloc((void **)&imgy_cuda, sizeof(int)*n);
    hipMalloc((void **)&imgcb_cuda, sizeof(int)*n);
    hipMalloc((void **)&imgcr_cuda, sizeof(int)*n);

    hipMemcpy(imgr_cuda, imgr, sizeof(int)*n, hipMemcpyHostToDevice);
    hipMemcpy(imgg_cuda, imgg, sizeof(int)*n, hipMemcpyHostToDevice);
    hipMemcpy(imgb_cuda, imgb, sizeof(int)*n, hipMemcpyHostToDevice);

    rgb2yuvKernel<<<blocksPerGrid,threadsPerBlock>>>(imgr_cuda, imgg_cuda, imgb_cuda, imgy_cuda, imgcb_cuda, imgcr_cuda, n);

    hipMemcpy(imgy, imgy_cuda, sizeof(int)*n, hipMemcpyDeviceToHost);
    hipMemcpy(imgcb, imgcb_cuda, sizeof(int)*n, hipMemcpyDeviceToHost);
    hipMemcpy(imgcr, imgcr_cuda, sizeof(int)*n, hipMemcpyDeviceToHost);

    hipFree(imgr_cuda);
    hipFree(imgg_cuda);
    hipFree(imgb_cuda);
    hipFree(imgy_cuda);
    hipFree(imgcb_cuda);
    hipFree(imgcr_cuda);
    printf("Printing... rgb2yuvKernel\n");
}

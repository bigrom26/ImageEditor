#include "hip/hip_runtime.h"
#include "cuda_methods.h"

#include <>
// CUDA runtime
#include <hip/hip_runtime.h>

// helper functions and utilities to work with CUDA
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include <hipfft/hipfft.h>
#include <stdlib.h>

void img2fft(unsigned char *src, unsigned char *dst, int w, int h){
    hipfftHandle plan;
    hipfftComplex *src_d, *src_h;
    hipfftComplex *dst_d, *dst_h;

    src_h = (hipfftComplex *) malloc(sizeof(hipfftComplex)*w*h);
    dst_h = (hipfftComplex *) malloc(sizeof(hipfftComplex)*w*h);

    for(int y=0; y<h; y++){
        for(int x=0; x<w; x++){
            src_h[y*w +x].x = (float)(src[y*w +x]);
            src_h[y*w +x].y = 0;
        }
    }

    checkCudaErrors(hipMalloc((void**)&src_d, sizeof(hipfftComplex)*w*h));
    checkCudaErrors(hipMalloc((void**)&dst_d, sizeof(hipfftComplex)*w*h));
    checkCudaErrors(hipMemcpy(src_d, src_h, h*w *sizeof(hipfftComplex), hipMemcpyHostToDevice));

    hipfftPlan2d(&plan, h, w, HIPFFT_C2C);
    hipfftExecC2C(plan, src_d, dst_d, HIPFFT_FORWARD);

    checkCudaErrors(hipMemcpy(dst_h, dst_d, h*w *sizeof(hipfftComplex), hipMemcpyDeviceToHost));

    for(int y=0; y<h; y++){
        for(int x=0; x<w; x++){
            const int luma32 = sqrt((dst_h[y*w +x].x)*(dst_h[y*w +x].x)+(dst_h[y*w +x].y)*(dst_h[y*w +x].y))/(1.0f*w);
            dst[y*w +x] = luma32 > 255 ? 255 : luma32 < 0 ? 0 : luma32;
        }
    }

    unsigned char *tmp = (unsigned char *)malloc(sizeof(unsigned char)*w/2);

    for(int i=0; i<h/2; i++){
        unsigned char* i_start = dst+i*w;
        unsigned char* i_end = dst + (i+h/2)*w + w/2;
        memcpy(tmp, i_start, sizeof(unsigned char)*w/2);
        memcpy(i_start, i_end, sizeof(unsigned char)*w/2);
        memcpy(i_end, tmp, sizeof(unsigned char)*w/2);
    }

    for(int i=0; i<h/2; i++){
        unsigned char* i_start = dst+i*w +w/2;
        unsigned char* i_end = dst + (i+h/2)*w;
        memcpy(tmp, i_start, sizeof(unsigned char)*w/2);
        memcpy(i_start, i_end, sizeof(unsigned char)*w/2);
        memcpy(i_end, tmp, sizeof(unsigned char)*w/2);
    }

    hipfftDestroy(plan);
    hipFree(src_d);
    hipFree(dst_d);
    free(src_h);
    free(dst_h);
    free(tmp);
}

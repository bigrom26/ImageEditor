#include "hip/hip_runtime.h"
#include <>
// CUDA runtime
#include <hip/hip_runtime.h>

// helper functions and utilities to work with CUDA
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

#include "test.h"

__global__ void vectorAdditionCUDA(const float* a, const float* b, float* c, int n)
{
    int ii = blockDim.x * blockIdx.x + threadIdx.x;
    if (ii < n)
        c[ii] = a[ii] + b[ii];
}

__global__ void cuprint()
{
    printf("Printing...\n");
}

void vectorAddition(const float* a, const float* b, float* c, int n) {

    float *a_cuda, *b_cuda, *c_cuda;
    unsigned int nBytes = sizeof(float) * n;
    int threadsPerBlock = 256;
    int blocksPerGrid   = (n + threadsPerBlock - 1) / threadsPerBlock;

    hipError_t err = hipSuccess;

    // allocate and copy memory into the device
    err = hipMalloc((void **)& a_cuda, nBytes);
    if (err != hipSuccess){
        fprintf(stderr, "Failed to allocate device vector A (error code %s) %d!\n", hipGetErrorString(err), err);
        exit(EXIT_FAILURE);
    }
    err = hipMalloc((void **)& b_cuda, nBytes);
    if (err != hipSuccess){
        fprintf(stderr, "Failed to allocate device vector B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipMalloc((void **)& c_cuda, nBytes);
    if (err != hipSuccess){
        fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipMemcpy(a_cuda, a, nBytes, hipMemcpyHostToDevice);
    if (err != hipSuccess){
        fprintf(stderr, "Failed to hipMemcpyHostToDevice vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipMemcpy(b_cuda, b, nBytes, hipMemcpyHostToDevice);
    if (err != hipSuccess){
        fprintf(stderr, "Failed to hipMemcpyHostToDevice vector B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    cuprint<<<1,1>>>();
    vectorAdditionCUDA<<<blocksPerGrid, threadsPerBlock>>>(a_cuda, b_cuda, c_cuda, n);
    // load the answer back into the host
    hipMemcpy(c, c_cuda, nBytes, hipMemcpyDeviceToHost);

    hipFree(a_cuda);
    hipFree(b_cuda);
    hipFree(c_cuda);
}
